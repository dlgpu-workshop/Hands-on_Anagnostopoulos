
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>

#define N 1

int main() {
    float h_F[N] = {5.0f};  // Host array for force
    float h_d[N] = {10.0f};  // Host array for displacement
    float *d_F, *d_d;  // Device arrays
    float result;  // Result of the dot product
    hipblasHandle_t handle;  // cuBLAS handle

    // Allocate device memory
    hipMalloc((void**)&d_F, N * sizeof(float));
    hipMalloc((void**)&d_d, N * sizeof(float));

    // Copy vectors to device
    hipMemcpy(d_F, h_F, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_d, h_d, N * sizeof(float), hipMemcpyHostToDevice);

    // Create cuBLAS handle
    hipblasCreate(&handle);

    // Calculate dot product
    hipblasSdot(handle, N, d_F, 1, d_d, 1, &result);

    // Print result
    printf("Work done: %f J\n", result);

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_F);
    hipFree(d_d);

    return 0;
}

