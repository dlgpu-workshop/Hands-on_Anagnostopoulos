#include <hipblas.h>
#include <hipDNN.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define M 3
#define K 2
#define N 4

void printMatrix(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%.2f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

// Function to initialize a matrix with random values between -1 and 1
void initializeMatrix(float* matrix, int rows, int cols) {
    srand(time(NULL));
    for (int i = 0; i < rows * cols; i++) {
        matrix[i] = (float)rand() / (float)(RAND_MAX / 2) - 1.0f;
    }
}

int main() {
    // Initialize cuBLAS and cuDNN
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);

    hipdnnHandle_t cudnnHandle;
    hipdnnCreate(&cudnnHandle);

    // Allocate and initialize input matrices on the device
    int rowsA = M, colsA = K, rowsB = K, colsB = N;
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc(&d_A, sizeof(float) * rowsA * colsA);
    hipMalloc(&d_B, sizeof(float) * rowsB * colsB);
    hipMalloc(&d_C, sizeof(float) * rowsA * colsB);

    // Initialize d_A and d_B with random values between -1 and 1
    float* h_A = (float*)malloc(sizeof(float) * rowsA * colsA);
    float* h_B = (float*)malloc(sizeof(float) * rowsB * colsB);
    initializeMatrix(h_A, rowsA, colsA);
    initializeMatrix(h_B, rowsB, colsB);
    hipMemcpy(d_A, h_A, sizeof(float) * rowsA * colsA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * rowsB * colsB, hipMemcpyHostToDevice);

    // Perform matrix multiplication using cuBLAS hipblasSgemm
    const float alpha = 1.0f;
    const float beta = 0.0f;
    ...

    // Print the result of the matrix multiplication
    float* h_C = (float*)malloc(sizeof(float) * rowsA * colsB);
    hipMemcpy(h_C, d_C, sizeof(float) * rowsA * colsB, hipMemcpyDeviceToHost);

    printf("A:\n");
    printMatrix(h_A, M, K);
    printf("\n"); 
    
    printf("B:\n");
    printMatrix(h_B, K, N);
    printf("\n"); 

    printf("C before ReLU activation:\n"); 
    printMatrix(h_C, M, N);
    printf("\n"); 

    // Create a cuDNN tensor descriptor for the output of the matrix multiplication
    hipdnnTensorDescriptor_t tensorDesc;
    hipdnnCreateTensorDescriptor(&tensorDesc);
    hipdnnSetTensor4dDescriptor(tensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, rowsA, colsB);

    // Create a cuDNN activation descriptor for the ReLU activation function
    hipdnnActivationDescriptor_t activationDesc;
    hipdnnCreateActivationDescriptor(&activationDesc);
    hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0);

    // Apply the ReLU activation function using cuDNN
    ...

    // Print the result after the ReLU activation
    hipMemcpy(h_C, d_C, sizeof(float) * rowsA * colsB, hipMemcpyDeviceToHost);

    printf("C after ReLU activation:\n");  
    printMatrix(h_C, M, N);

    // Free the memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}